#include "hip/hip_runtime.h"
/**
 * @file test_bp.cu
 *
 * Test program for belief propagation
*/

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>
#include <fstream>

#include <time.h>

#include <gunrock/gunrock.h>
#include <gunrock/csr.cuh>
#include <gunrock/graphio/market.cuh>

#include <moderngpu.cuh>


int RunBP(std::string in_edges_filename, std::string in_nodes_filename, std::string out_filename) {
    clock_t start, end;
    double time_elapsed;

    typedef int VertexId;
    typedef float Value;
    typedef int SizeT;

    gunrock::Csr<VertexId, SizeT , Value> csr(false);
    if(gunrock::graphio::BuildMarketGraph_BP<true, VertexId, SizeT, Value>((char *)in_edges_filename.c_str(), (char *)in_nodes_filename.c_str(), csr, false, false, false) != 0) {
        perror("Unable to build csr...exiting\n");
        return 1;
    }

    GRTypes data_t;
    data_t.VTXID_TYPE = VTXID_INT;
    data_t.SIZET_TYPE = SIZET_INT;
    data_t.VALUE_TYPE = VALUE_FLOAT;

    GRGraph *graphi = new GRGraph();
    GRGraph *grapho = new GRGraph();

    graphi->num_nodes = csr.nodes;
    graphi->num_edges = csr.edges;
    graphi->row_offsets = csr.row_offsets;
    graphi->col_indices = csr.column_indices;
    graphi->edge_values = csr.edge_values;
    graphi->node_value1 = csr.node_values;

    GRSetup *config = InitSetup(1, {0});
    config->quiet = false;

    start = clock();

    gunrock_bp(grapho, graphi, config, data_t);

    end = clock();
    time_elapsed = (double)(end - start)/(CLOCKS_PER_SEC);

    std::ofstream out;
    out.open(out_filename.c_str());

    printf("Nodes\tEdges\tTime(s)\n");
    out << "Nodes,Edges,Time(s)" << std::endl;
    printf("%d\t%d\t%.6f\n", csr.nodes, csr.edges, time_elapsed);
    out << csr.nodes << "," << csr.edges << "," << time_elapsed << std::endl;
    out.close();

    float *beliefs = new float[graphi->num_nodes];
    memcpy(beliefs, grapho->node_value1, graphi->num_nodes);

    for(int node = 0; node < graphi->num_nodes; ++node) {
        printf("Belief [%.10e]\n", beliefs[node]);
    }

    if(graphi) {
        delete graphi;
    }
    if(grapho) {
        delete grapho;
    }
    if(beliefs) {
        delete[] beliefs;
    }

    return 0;
}

int main(int argc, char** argv)
{

    RunBP("/home/mjt5v/Source_Code/gunrock/tests/bp/test.bif.edges.mtx", "/home/mjt5v/Source_Code/gunrock/tests/bp/test.bif.nodes.mtx", "gunrock_bp.csv");

    return 0;
}